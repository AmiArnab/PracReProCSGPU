
#include "hip/hip_runtime.h"

#include "hip/hip_runtime_api.h"
#include "hipsolver.h"

#include <cstdio>
#include <cstdlib>
#include <iostream>

using namespace std;

hipError_t cudaStatus;

void printError(float pos)
{
	if (cudaStatus != hipSuccess)
	{
		cout << "Error! " << pos << "\nEnter any key to continue...";
		getchar();
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) cout << "Device reset not successful!\nExiting...\n";
		exit(EXIT_FAILURE);
	}
	else
	{
		cout << "Success! " << pos << endl;
	}
}

void ExitProgram()
{
	cudaStatus = hipDeviceReset();
	printError(-1);
	exit(EXIT_FAILURE);
}

int main()
{
	hipsolverHandle_t my_handle;
	hipsolverDnCreate(&my_handle);

	const int M = 3;
	const int N = 3;
	float *host_A = NULL;
	float *host_S = NULL;

	float *dev_A = NULL;
	float *dev_S = NULL;
	float *dev_U = NULL;
	float *dev_V = NULL;

	float *work = 0;
	int work_size = 0;
	int *dev_info = 0;

	int rank = 0;
	float threshold = 0.001;

	cout << "Starting program...\n";

	host_A = new float[M*N*sizeof(float)];
	host_S = new float[M*sizeof(float)];

	for (unsigned int i = 0; i < M; ++i)
	{
		for (unsigned int j = 0; j < N; ++j)
		{
			*(host_A + (i*M) + j) = i*j + 1;
		}
	}

	for (unsigned int i = 0; i < M; ++i)
	{
		for (unsigned int j = 0; j < N; ++j)
		{
			cout << *(host_A + (i*M) + j) << " ";
		}
		cout << endl;
	}

	cout << "Host allocation and initialization finished!\nStarting GPU allocation and copy...\n";

	cudaStatus = hipSetDevice(0);
	printError(0);

	cudaStatus = hipMalloc((void**)&dev_A, M * N * sizeof(float));
	printError(1.1);
	cudaStatus = hipMalloc((void**)&dev_S, M * sizeof(float));
	printError(1.2);
	cudaStatus = hipMalloc((void**)&dev_U, M * M * sizeof(float));
	printError(1.3);
	cudaStatus = hipMalloc((void**)&dev_V, N * N * sizeof(float));
	printError(1.4);

	cudaStatus = hipMemcpy(dev_A, host_A, M*N*sizeof(float), hipMemcpyHostToDevice);
	printError(2);

	hipsolverStatus_t cusolverstatus;

	cusolverstatus = hipsolverDnSgesvd_bufferSize(my_handle, M, N, &work_size);
	if (cusolverstatus != HIPSOLVER_STATUS_SUCCESS) { cout << "cuSolver buffer allocation failed!\n";  hipDeviceReset(); exit(EXIT_FAILURE);}

	cudaStatus = hipMalloc((void**)&work, work_size * sizeof(float));
	printError(3.1);
	cudaStatus = hipMalloc((void**)&dev_info, sizeof(int));
	printError(3.2);

	cusolverstatus = hipsolverDnSgesvd(my_handle, 'A', 'A', M, N, dev_A, M, dev_S, dev_U, M, dev_V, N, work, work_size, NULL, dev_info);
	cudaStatus = hipDeviceSynchronize();
	if (cusolverstatus != HIPSOLVER_STATUS_SUCCESS) { cout << "SVD failed!\n"; hipDeviceReset(); exit(EXIT_FAILURE);}
	printError(4.1);

	cout << "SVD completed!\n";

	cudaStatus = hipMemcpy(host_S,dev_S,M*sizeof(float),hipMemcpyDeviceToHost);
	printError(4.2);

	cudaStatus = hipFree(dev_A);
	printError(5.1);
	cudaStatus = hipFree(dev_S);
	printError(5.2);
	cudaStatus = hipFree(dev_U);
	printError(5.3);

    cudaStatus = hipDeviceReset();
	printError(6);

	for (unsigned int i = 0; i < M; ++i)
	{
		if (*(host_S + i) > threshold) ++rank;
	}

	cout << "Singular value threshold for computing rank: " << threshold << endl;
	cout << "Rank: " << rank << endl;

	delete [] host_A;
	delete [] host_S;

	cout << "Enter anything to exit...\n";
	getchar();

    return 0;
}
