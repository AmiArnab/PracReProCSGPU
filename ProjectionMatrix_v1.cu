#define IDX2C(i,j,ld) (((j)*(ld))+(i))

#include "hip/hip_runtime.h"

#include "hipblas.h"

#include <iostream>
#include <cstdio>
using namespace std;

int getProjectionMatrix(float *Phi, float *P, int M, int N)
{
	float *dev_Phi = NULL;
	float *dev_P = NULL;
	float *dev_I = NULL;
	float *host_I = NULL;
	float *host_P = NULL;
	float *host_Phi = NULL;
	float alpha = 1;
	float beta = 0;
	float gamma = -1;

	hipblasHandle_t chandle;
	hipblasOperation_t topA = HIPBLAS_OP_N;
	hipblasOperation_t topB = HIPBLAS_OP_T;

	hipError_t cerr;

	host_I = new float[M*M];
	host_P = new float[M*N];
	host_Phi = new float[M*M];

	cerr = hipMalloc((void **)&dev_Phi, M*M*sizeof(float));
	cerr = hipMalloc((void **)&dev_P, M*N*sizeof(float));
	cerr = hipMalloc((void **)&dev_I, M*M*sizeof(float));

	for (int i = 0; i < M; ++i)
	{
		for (int j = 0; j < M; ++j)
		{
			host_I[IDX2C(i,j,M)] = (i==j)?1:0;
		}
	}

	for (int i = 0; i < M; ++i)
	{
		for (int j = 0; j < M; ++j)
		{
			cout << host_I[IDX2C(i, j, M)] << " ";
		}
		cout << endl;
	}

	for (int i = 0; i < M; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			host_P[IDX2C(i, j, M)] = P[(i*M) + j];
		}
	}

	for (int i = 0; i < M; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			cout << host_P[IDX2C(i, j, M)] << " ";
		}
		cout << endl;
	}

	hipblasCreate(&chandle);

	hipblasSetMatrix(M, N, sizeof(float), P, M, dev_P, M);
	hipblasSetMatrix(M, M, sizeof(float), Phi, M, dev_Phi, M);
	hipblasSetMatrix(M, M, sizeof(float), host_I, M, dev_I, M);

	hipblasSgemm(chandle, topA, topB, M, M, N,&alpha,dev_P,M,dev_P,M,&beta,dev_Phi,M);
	hipblasSgeam(chandle, topA, topA, M, N, &alpha, dev_I, M, &gamma, dev_Phi, M, dev_Phi, M);

	hipblasGetMatrix(M, M, sizeof(float), dev_Phi, M, host_Phi, M);

	for (int i = 0; i < M; ++i)
	{
		for (int j = 0; j < M; ++j)
		{
			Phi[(i*M) + j] = host_Phi[IDX2C(i, j, M)];
		}
	}

	hipFree(dev_Phi);
	hipFree(dev_P);

	delete[] host_I;
	delete[] host_P;
	delete[] host_Phi;

	return 0;
}

int main()
{
	int M = 3;
	int N = 3;
	float *arr = NULL;
	float *phi = NULL;

	arr = new float[M*N];
	phi = new float[M*M];

	for (int i = 0; i < M; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			arr[(i*M)+j] = (i==j)?1:0;
		}
	}

	getProjectionMatrix(phi, arr, M, N);

	for (int i = 0; i < M; ++i)
	{
		for (int j = 0; j < M; ++j)
		{
			cout << phi[(i*M) + j] << " ";
		}
		cout << endl;
	}

	getchar();
	delete[] arr;
    return 0;
}