#include "getProjectionMatrix.h"

int getProjectionMat(float *dev_Phi, float *dev_P, int M, int N)
{
	float alpha = 1;
	//float beta = 0;
	float gamma = -1;
	//float *dev_I = NULL;

	hipblasHandle_t chandle;
	hipblasOperation_t topA = HIPBLAS_OP_N;
	hipblasOperation_t topB = HIPBLAS_OP_T;

	//hipMalloc((void **)&dev_I, M*M*sizeof(float));
	thrust::device_vector<float> devI(M*M);
	thrust::fill(devI.begin(),devI.end(),0);
	for(int i=0;i<M;++i)
	{
		devI[i*M+i] = 1;
	}

	hipblasCreate(&chandle);
	hipblasSgemm(chandle, topA, topB, M, M, N,&gamma,dev_P,M,dev_P,M,&alpha,thrust::raw_pointer_cast(&devI[0]),M);
	hipMemcpy(dev_Phi,thrust::raw_pointer_cast(&devI[0]),M*M*sizeof(float),hipMemcpyDeviceToDevice);

	//hipblasSgemm(chandle, topA, topB, M, M, N,&alpha,dev_P,M,dev_P,M,&beta,dev_Phi,M);
	//hipblasSgeam(chandle, topA, topA, M, N, &alpha, thrust::raw_pointer_cast(&devI[0]), M, &gamma, dev_Phi, M, dev_Phi, M);

	hipblasDestroy(chandle);

	return 0;
}
