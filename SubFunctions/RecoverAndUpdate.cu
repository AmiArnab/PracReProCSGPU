#include "RecoverAndUpdate.h"

int UpdateSubspace(thrust::device_vector<float> *devPt, float *dev_Lt, thrust::device_vector<float> *devPLt, int M, int N, int d, int rank, bool dosvd)
{
	thrust::device_vector<float> devLt(dev_Lt, dev_Lt + M);
	thrust::device_vector<float>::iterator it = (*devPLt).begin();
	thrust::device_vector<float>::iterator Lit = devLt.begin();
	(*devPLt).insert(it, it + M, it - M);
	(*devPLt).insert(it - M, Lit, Lit + M - 1);

	if (dosvd == true)
	{
		//GetApproxBasisUpdate(devPt, thrust::raw_pointer_cast(&(*devPLt)[0]),M,N,rank);
		hipsolverHandle_t my_handle;
		hipsolverDnCreate(&my_handle);

		float *dev_S = NULL;
		float *dev_U = NULL;
		float *dev_V = NULL;

		float *work = 0;
		int work_size = 0;
		int *dev_info = 0;

		thrust::device_vector<float> devU(M*M);

		hipMalloc((void**)&dev_S, M * sizeof(float));
		hipMalloc((void**)&dev_U, M * M * sizeof(float));
		hipMalloc((void**)&dev_V, N * N * sizeof(float));

		hipsolverDnSgesvd_bufferSize(my_handle, M, N, &work_size);
		hipMalloc((void**)&work, work_size * sizeof(float));
		hipMalloc((void**)&dev_info, sizeof(int));
		hipsolverDnSgesvd(my_handle, 'A', 'A', M, N, thrust::raw_pointer_cast(&(*devPLt)[0]), M, dev_S, dev_U, M, dev_V, N, work, work_size, NULL, dev_info);
		hipDeviceSynchronize();

		float const alpha(1.0);
		float const beta(0.0);
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, M, &alpha, dev_U, M, &beta, NULL, M, thrust::raw_pointer_cast(&devU[0]), M);
		hipblasDestroy(handle);
		hipDeviceSynchronize();

		for (int i = 0; i < M; ++i)
		{
			for (int j = 0; j < rank; ++j)
			{
				(*devPt)[i*rank + j] = devU[i*M + j];
			}
		}


		hipFree(dev_V);
		hipFree(dev_S);
		hipFree(dev_U);

		hipsolverDnDestroy(my_handle);
	}

	return 0;
}

