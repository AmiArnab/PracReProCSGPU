#include "getPruneVector.h"

int Prune(unsigned int *T, float *St, int M, unsigned int s)
{
	thrust::device_vector<float> pkeys(St,St+M-1);
	thrust::host_vector<int> skeys(M);
	thrust::counting_iterator<int> it(0);
	thrust::device_vector<int> indices(M);
	thrust::copy(it, it + indices.size(), indices.begin());
	thrust::sort_by_key(pkeys.begin(), pkeys.end(), indices.begin());
	thrust::copy(indices.begin(), indices.end(), skeys.begin());
	for (int i = 0; i < M; i++)
	{
		T[i] = 0;
	}
	for (int i = M; i>M - s; i--)
	{
		T[skeys[i]] = 1;
	}

	return 0;
}
