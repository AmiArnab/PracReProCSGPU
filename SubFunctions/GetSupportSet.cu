#include "hip/hip_runtime.h"
#include "GetSupportSet.h"

int getSupportSet(int *supportset, float *inputarr, int N)
{
	for (int i = 0; i < N; ++i)
	{
		if (inputarr[i] != 0.0) supportset[i] = 1;
		else supportset[i] = 0;
	}
	return 0;
}

float findIntersection(int *Told1, int *Told2, int M)
{
	int intersectno = 0;
	int cardinality = 0;
	for (int i = 0; i < M; ++i)
	{
		if (Told2[i] == 1) cardinality++;
		if (Told1[i] == Told2[i]) intersectno++;
	}
	return (float)(intersectno / cardinality);
}

float getSetDifference(int *Told1, int *Told2, int M)
{
	int diffcount = 0;
	int totalcount = 0;
	for (int i = 0; i < M; ++i)
	{
		if (Told1[i] == 1) totalcount++;
		if ((Told2[i] == 1) && (Told1[i] == 0)) diffcount++;
	}
	return (float)(diffcount / totalcount);
}

int getSupportCardinality(int *T, int M)
{
	int count = 0;
	for (int i = 0; i < M; ++i)
	{
		if (T[i] == 1)count++;
	}
	return count;
}
float computeOmega(float *dev_Mt, int M)
{
	hipblasHandle_t chandle;
	float host_omega;
	hipblasCreate(&chandle);
	//float *dev_Mt;
	//hipMalloc((void**)&dev_Mt, M*sizeof(float));
	//hipMemcpy(dev_Mt, Mt, M*sizeof(float), hipMemcpyDeviceToDevice);
	hipblasSdot(chandle, M, dev_Mt, 1, dev_Mt, 1, &host_omega);
	//hipFree(dev_Mt);
	host_omega = host_omega / M;
	host_omega = sqrtf(host_omega);

	hipblasDestroy(chandle);

	return host_omega;
}
int computeLt(float *dev_Lt, float *dev_Mt, float *dev_St, int M)
{
	thrust::device_vector<float> devLt(dev_Lt,dev_Lt+M-1);
	thrust::device_vector<float> devMt(dev_Mt,dev_Mt+M-1);
	thrust::device_vector<float> devSt(dev_St,dev_St+M-1);
	for (int i = 0; i < M; i++)
	{
		devLt[i] = devMt[i] - devSt[i];
	}
	return 0;
}
