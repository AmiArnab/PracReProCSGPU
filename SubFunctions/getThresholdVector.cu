#include "hip/hip_runtime.h"
#include "getThresholdVector.h"

int Thresh(unsigned int *T, float *dev_St, int M,float omega)
{
        thrust::device_vector<float> rsvectors(dev_St,dev_St+M);
	for (int i = 0; i < M; ++i)
	{
		if (abs(rsvectors[i]) >= omega)
		{
			T[i] = 1;
		}
		else
		{
			T[i] = 0;
		}
	}
	return 0;
}
