#include "ConcatenateVecHorizontal.h"

int ConcatenateMatrix(float **dev_outmat, float *dev_inmat, float **dev_listmat, int M, int N, int n)
{
	float *dev_newout, *dout;
	hipMalloc((&dev_newout), (N + n)*M*sizeof(float));
	hipMemset(dev_newout, 0, (N + n)*M*sizeof(float));
	thrust::device_vector<float> dinvec(dev_inmat, dev_inmat + (M*N));
	thrust::device_vector<float> doutvec(dev_newout, dev_newout + (N + n)*M);
	for (int j = 0; j < N; ++j)
	{
		for (int i = 0; i < M; ++i)
		{
			doutvec[(i*(N + n)) + j] = dinvec[(i*N) + j];
		}
	}

	for (int k = 0; k < n; k++)
	{
		thrust::device_vector<float> dlsvec(*(dev_listmat+k), *(dev_listmat +k) + M);
		for (int j = N; j < (N + n); ++j)
		{
			for (int i = 0; i < M; ++i)
			{
				doutvec[(i*(N + n)) + j] = dlsvec[i];
			}
		}
	}

	dout = thrust::raw_pointer_cast(doutvec.data());
	hipMemcpy(*dev_outmat, dout, M*(N + n)*sizeof(float), hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
	hipFree(dev_newout);
	return 0;
}
