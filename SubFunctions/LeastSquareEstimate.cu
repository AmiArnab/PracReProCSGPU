#include "LeastSquareEstimate.h"

int getLeastSquareEstimate(float *Stadd, float *yt, float *Phit, unsigned int *Tt, int M, int N)
{
	float *dev_yt = NULL;
	float *dev_Phit = NULL;
	float *dev_tempA = NULL;

	float *dev_auxArray = NULL;
	const float *dev_Array[1] = {dev_auxArray}; //Change this
	float *dev_ArrayLU[1] = { dev_auxArray };
	int *dev_pivotarrr = NULL;
	int *dev_infoarr = NULL;
	float *dev_Carray[1]; //Change this

	//hipError_t error;
	hipblasHandle_t bhandle;
	hipsolverHandle_t shandle;
	hipsolverDnCreate(&shandle);
	//hipsolverStatus_t status;
	hipblasOperation_t opN = HIPBLAS_OP_N;
	hipblasOperation_t opT = HIPBLAS_OP_T;
	float alpha = 1, gamma = 0;//, beta = 1;

	hipblasCreate(&bhandle);

	hipMalloc((void**)&dev_yt, M*sizeof(float));
	hipMalloc((void**)&dev_Phit, M*M*sizeof(float));
	hipMalloc((void**)&dev_tempA, M*M*sizeof(float));

	hipMalloc((void***)&dev_auxArray, M*M*sizeof(float*));
	hipMalloc((void**)&dev_pivotarrr, M*sizeof(int));
	hipMalloc((void**)&dev_infoarr, sizeof(int));
	hipMalloc((void***)&dev_Carray, sizeof(float*));

	hipMemcpy(dev_yt,yt,M*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_Phit, Phit, M*M*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(dev_tempA, Phit, M*M*sizeof(float), hipMemcpyDeviceToHost);

	//dev_Array[0] = dev_auxArray;

	//Convert dev_Phit to column major format first
	hipblasSgemm(bhandle,opT,opN,M,M,M,&alpha,dev_Phit,M,dev_Phit,M,dev_tempA,&gamma,M);

	//Invert Matrix
	hipMemcpy(dev_auxArray, dev_tempA, M*M*sizeof(float), hipMemcpyDeviceToDevice);

	hipblasSgetrfBatched(bhandle,M,dev_ArrayLU,M,dev_pivotarrr,dev_infoarr,1); //Column Major format
	//hipblasSgetriBatched(bhandle, M, dev_Array, M, dev_pivotarrr, dev_Carray, M, dev_infoarr, 1);
	hipblasSgetriBatched(bhandle, M, dev_Array, M, dev_pivotarrr, dev_Carray, M, dev_infoarr, 1);

	hipMemcpy(dev_tempA, *dev_Array, M*M*sizeof(float), hipMemcpyDeviceToDevice);

	hipblasSgemm(bhandle, opN, opT, M, M, M, &alpha, dev_tempA, M, dev_Phit, M,NULL, &gamma,M);
	hipblasSgemv(bhandle, opN, M, M, &alpha, dev_tempA, M, dev_yt, 1, &gamma, NULL, 1);

	hipblasGetVector(M, sizeof(float), Stadd, 1, dev_yt, 1);

	hipblasDestroy(bhandle);

	hipFree(dev_yt);
	hipFree(dev_Phit);
	hipFree(dev_tempA);
	//hipFree(dev_Array);
	hipFree(dev_pivotarrr);
	hipFree(dev_infoarr);
	//hipFree(dev_Carray);

	return 0;
}
