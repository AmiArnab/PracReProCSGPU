#include "getApproxBasisUpdate.h"
int GetApproxBasisUpdate(thrust::device_vector<float> *appbase, float *dev_Mtrain, int M, int N, int r)
{
	hipsolverHandle_t my_handle;
	hipsolverDnCreate(&my_handle);

	float *dev_S = NULL;
	float *dev_U = NULL;
	float *dev_V = NULL;

	float *work = 0;
	int work_size = 0;
	int *dev_info = 0;

	thrust::device_vector<float> devU(M*M);

	hipMalloc((void**)&dev_S, M * sizeof(float));
	hipMalloc((void**)&dev_U, M * M * sizeof(float));
	hipMalloc((void**)&dev_V, N * N * sizeof(float));

	hipsolverDnSgesvd_bufferSize(my_handle, M, N, &work_size);
	hipMalloc((void**)&work, work_size * sizeof(float));
	hipMalloc((void**)&dev_info, sizeof(int));
	hipsolverDnSgesvd(my_handle, 'A', 'A', M, N, dev_Mtrain, M, dev_S, dev_U, M, dev_V, N, work, work_size, NULL, dev_info);
	hipDeviceSynchronize();
	
	float const alpha(1.0);
	float const beta(0.0);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, M, M, &alpha, dev_U, M, &beta, NULL, M, thrust::raw_pointer_cast(&devU[0]), M);
	hipblasDestroy(handle);
	hipDeviceSynchronize();

	for (int i = 0; i < M; ++i)
	{
		for (int j = 0; j < r; ++j)
		{
			(*appbase)[i*r + j] = devU[i*M + j];
		}
	}


	hipFree(dev_V);
	hipFree(dev_S);
	hipFree(dev_U);

	hipsolverDnDestroy(my_handle);

	return 0;
}
