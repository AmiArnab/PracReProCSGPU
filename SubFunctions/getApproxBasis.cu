#include "getApproxBasis.h"

int GetApproxBasis(float *dev_approx_basis, int *Ma, int *Na, float *dev_Mtrain, int M, int N, float b)
{
	hipsolverHandle_t my_handle;
	hipsolverDnCreate(&my_handle);

	//int Malocal = 0;
	//int Nalocal = 0;
	int maxindex = 0;
	float sum = 0, maxsum = 0;

	float *dev_S = NULL;
	float *dev_U = NULL;
	float *dev_V = NULL;

	float *work = 0;
	int work_size = 0;
	int *dev_info = 0;

	hipMalloc((void**)&dev_S, N * sizeof(float));
	hipMalloc((void**)&dev_U, M * M * sizeof(float));
	hipMalloc((void**)&dev_V, N * N * sizeof(float));

	//hipsolverStatus_t cusolverstatus; // Not used

	hipsolverDnSgesvd_bufferSize(my_handle, M, N, &work_size);
	hipMalloc((void**)&work, work_size * sizeof(float));
	hipMalloc((void**)&dev_info, sizeof(int));

	hipsolverDnSgesvd(my_handle, 'A', 'A', M, N, dev_Mtrain, M, dev_S, dev_U, M, dev_V, N, work, work_size, NULL, dev_info);
	hipDeviceSynchronize();
	//hipsolverDnDestroy(my_handle);
	thrust::device_vector<float> svalues(dev_S,dev_S+N);
	for(int i=0;i<N;++i) //changed to N
	{
	        maxsum+=svalues[i];
	}

        for(int i=0;i<N;++i) //changed to N
	{
	        sum+=svalues[i];
	        if((sum/maxsum)>b)
	        {
	                maxindex = i;
	                break;
	        }
	}

	hipMalloc((void**)&dev_approx_basis, M * maxindex * sizeof(float));
	hipMemset(dev_approx_basis,0,M*maxindex*sizeof(float));
	thrust::device_vector<float> rsvectors(dev_U,dev_U+(M*M));
	//thrust::device_vector<float> apbvectors(dev_approx_basis,dev_approx_basis+(M*maxindex));
	thrust::device_vector<float> apbvectors(M*maxindex);
	thrust::copy(rsvectors.begin(), rsvectors.begin()+(M*maxindex), apbvectors.begin());
	hipMemcpy(dev_approx_basis, thrust::raw_pointer_cast(&apbvectors[0]), M*maxindex*sizeof(float), hipMemcpyDeviceToDevice);
	*Ma = M;
	*Na = maxindex;

	hipsolverDnDestroy(my_handle);
        
	hipFree(dev_S);
	hipFree(dev_U);
	hipFree(dev_V);
	
	return 0;
}
