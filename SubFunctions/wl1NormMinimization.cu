#include "wl1NormMinimization.h"

int performWtdBregmanIterRegularization(float *dev_x, float *dev_yt, float *dev_Phit, int *T, float lambda, float epsilon, int M, int N)
{
	float err = epsilon + 100;
	float mu = 1;
	float delta = 0.7;

	float *dev_temp1 = NULL;
	float *dev_currv = NULL;
	float *dev_u = NULL;
	float *dev_tempy = NULL;
	float *dev_tempx = NULL;

	hipblasHandle_t handle;
	hipblasOperation_t opN = HIPBLAS_OP_N;
	hipblasOperation_t opT = HIPBLAS_OP_T;

	float alpha1 = 1;
	float alpha2 = -1;
	float beta1 = 1;
	//float beta2 = -1;

	hipblasCreate(&handle);

	hipMalloc((void **)&dev_u, M*sizeof(float));
	hipMalloc((void **)&dev_currv, M*sizeof(float));
	hipMalloc((void **)&dev_temp1, M*sizeof(float));
	hipMalloc((void **)&dev_tempy, M*sizeof(float));
	hipMalloc((void **)&dev_tempx, M*sizeof(float));

	thrust::device_vector<float> devyt(dev_yt, dev_yt + M - 1);
	thrust::device_vector<float> devx(dev_x, dev_x + M - 1);
	thrust::device_vector<float> devu(dev_u, dev_u + M - 1);
	thrust::device_vector<float> devcurrv(dev_currv, dev_currv + M - 1);
	thrust::device_vector<float> devtemp1(dev_temp1, dev_temp1 + M - 1);
	thrust::device_vector<float> devtempy(dev_tempy, dev_tempy + M - 1);
	thrust::device_vector<float> devtempx(dev_tempx, dev_tempx + M - 1);

	for (int i = 0; i<M; ++i)
	{
		devtempx[i] = 0;
		devu[i] = 0;
		devcurrv[i] = 0;
		devtempy[i] = 0;
	}


	while (err > epsilon)
	{
		for (int i = 0; i<M; ++i)
		{
			if (T[i] == 1)
			{
				devtempx[i] = lambda*devu[i];
			}
			else
			{
				devtempx[i] = devu[i];
			}
			
		}
		hipblasSgemv(handle, opN, M, M, &alpha2, dev_Phit, M, thrust::raw_pointer_cast(&devtempx[0]), 1, &beta1, thrust::raw_pointer_cast(&devtempy[0]), 1);
		hipblasSgemv(handle, opT, M, M, &alpha1, dev_Phit, M, thrust::raw_pointer_cast(&devtempy[0]), 1, &beta1, thrust::raw_pointer_cast(&devcurrv[0]), 1);

		for (int i = 0; i < M; ++i)
		{
			if (devcurrv[i] > mu)
			{
				devu[i] = delta*(devcurrv[i] - mu);
			}
			else if (devcurrv[i] < -mu)
			{
				devu[i] = delta*(devcurrv[i] + mu);
			}
			else
			{
				devu[i] = 0;
			}
		}

		thrust::copy(devyt.begin(), devyt.end(), devtempy.begin());
		hipblasSgemv(handle, opN, M, M, &alpha2, dev_Phit, M, thrust::raw_pointer_cast(&devu[0]), 1, &beta1, thrust::raw_pointer_cast(&devtempy[0]), 1);

		hipblasSnrm2(handle, M, thrust::raw_pointer_cast(&devtempy[0]), 1, &err);
	}

	hipblasDestroy(handle);

	hipFree(dev_u);
	hipFree(dev_tempy);
	hipFree(dev_currv);
	hipFree(dev_temp1);
	hipFree(dev_tempx);

	return 0;
}
