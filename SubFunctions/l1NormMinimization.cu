#include "l1NormMinimization.h"

int performBregmanIterRegularization(float *dev_x, float *dev_yt, float *dev_Phit, float epsilon, int M, int N)
{
	float err = epsilon + 100;
	float mu = 1;
	float delta = 0.7;

	float *dev_temp1 = NULL;
	float *dev_currv = NULL;
	float *dev_u = NULL;
	float *dev_tempy = NULL;

	hipblasHandle_t handle;
	hipblasOperation_t opN = HIPBLAS_OP_N;
	hipblasOperation_t opT = HIPBLAS_OP_T;

	float alpha1 = 1;
	float alpha2 = -1;
	float beta1 = 1;
	//float beta2 = -1;

	hipMalloc((void **)&dev_u, M*sizeof(float));
	hipMalloc((void **)&dev_currv, M*sizeof(float));
	hipMalloc((void **)&dev_temp1, M*sizeof(float));
	hipMalloc((void **)&dev_tempy, M*sizeof(float));

	thrust::device_vector<float> devyt(dev_yt, dev_yt + M - 1);
	thrust::device_vector<float> devx(dev_x, dev_x + M - 1);
	thrust::device_vector<float> devu(dev_u, dev_u + M - 1);
	thrust::device_vector<float> devcurrv(dev_currv, dev_currv + M - 1);
	thrust::device_vector<float> devtemp1(dev_temp1, dev_temp1 + M - 1);
	thrust::device_vector<float> devtempy(dev_tempy, dev_tempy + M - 1);
	

	while (err > epsilon)
	{
		//hipblasSgemv(handle, opN, M, M, &alpha2, dev_Phit, M, thrust::raw_pointer_cast(&devu[0]), 1, &beta1, thrust::raw_pointer_cast(&devtempy[0]), 1);
		hipblasSgemv(handle, opT, M, M, &alpha1, dev_Phit, M, thrust::raw_pointer_cast(&devtempy[0]), 1, &beta1, thrust::raw_pointer_cast(&devcurrv[0]), 1);

		for (int i = 0; i < M; ++i)
		{
			if (devcurrv[i] > mu)
			{
				devu[i] = delta*(devcurrv[i] - mu);
			}
			else if (devcurrv[i] < -mu)
			{
				devu[i] = delta*(devcurrv[i] + mu);
			}
			else
			{
				devu[i] = 0;
			}
		}

		thrust::copy(devyt.begin(), devyt.end(), devtempy.begin());
		hipblasSgemv(handle, opN, M, M, &alpha2, dev_Phit, M, thrust::raw_pointer_cast(&devu[0]), 1, &beta1, thrust::raw_pointer_cast(&devtempy[0]), 1);

		hipblasSnrm2(handle, M, thrust::raw_pointer_cast(&devtempy[0]), 1, &err);
	}

	hipblasDestroy(handle);

	hipFree(dev_u);
	hipFree(dev_tempy);
	hipFree(dev_currv);
	hipFree(dev_u);

	return 0;
}
