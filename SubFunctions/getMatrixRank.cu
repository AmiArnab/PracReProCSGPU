#include "getMatrixRank.h"

int getMatRank(int *matrank, float *dev_Mt, float threshold, int M, int N)
{
	hipsolverHandle_t my_handle;
	hipsolverDnCreate(&my_handle);

	float *dev_S = NULL;
	float *dev_U = NULL;
	float *dev_V = NULL;

	float *work = 0;
	int work_size = 0;
	int *dev_info = 0;
	int info = 0;
	
	hipMalloc((void**)&dev_S, N * sizeof(float));
	hipMalloc((void**)&dev_U, M * M * sizeof(float));
    	hipMalloc((void**)&dev_V, N * N * sizeof(float));

	hipsolverDnSgesvd_bufferSize(my_handle, M, N, &work_size);
	
	hipMalloc((void**)&work, work_size * sizeof(float));
	hipMalloc((void**)&dev_info, sizeof(int));
	cout << "Before SVD\n";
	hipsolverStatus_t stat = hipsolverDnSgesvd(my_handle, 'A', 'A', M, N, dev_Mt, M, dev_S, dev_U, M, dev_V, N, work, work_size, NULL, dev_info);
	//hipsolverStatus_t stat = hipsolverDnSgesvd(my_handle, 'A', 'A', M, N, dev_Mt, M, dev_S, dev_U, M, NULL, N, work, work_size, NULL, dev_info);
	switch(stat)
	{
		case HIPSOLVER_STATUS_SUCCESS:           std::cout << "SVD computation success\n";                       break;
		case HIPSOLVER_STATUS_NOT_INITIALIZED:   std::cout << "Library cuSolver not initialized correctly\n";    break;
		case HIPSOLVER_STATUS_INVALID_VALUE:     std::cout << "Invalid parameters passed\n";                     break;
		case HIPSOLVER_STATUS_INTERNAL_ERROR:    std::cout << "Internal operation failed\n";                     break;
	}

	hipDeviceSynchronize();
	cout << "After SVD\n";
        //thrust::device_vector<float> sigvals(dev_S,dev_S+N);
	//thrust::host_vector<float> tempval(N);
	//hipMemcpy(thrust::raw_pointer_cast(&tempval[0]),dev_S,N*sizeof(float),hipMemcpyDeviceToHost);
	//thrust::device_vector<float> sigs = tempval;
	//hipMemcpy(&info,dev_info,sizeof(int),hipMemcpyDeviceToHost);
	//cout << info << endl;
	/*hipMemcpy(thrust::raw_pointer_cast(&sigvals[0]),dev_S,N*sizeof(float),hipMemcpyDeviceToDevice);
        cout << "After Vector\n";
	for(int i=0;i<N;++i)
	{
	        if(sigvals[i] > threshold)
	        {
	                (*matrank)++;
	        }
	}*/
        cout << "After rank\n";
	hipsolverDnDestroy(my_handle);
	hipFree(dev_V);
	hipFree(dev_S);
	hipFree(dev_U);
	hipFree(work);
	hipFree(dev_info);
	
        return 0;
}
