
#include "hip/hip_runtime.h"


#include <iostream>
#include <cstdio>
using namespace std;

int getSupportSet(int *supportset, float *inputarr, int N)
{
	for (int i = 0; i < N; ++i)
	{
		if (inputarr[i] != 0.0) supportset[i] = 1;
		else supportset[i] = 0;
	}
	return 0;
}

int main()
{
	int N = 10;
	int *sup = new int[N];
	float *arr = new float[N];

	for (int i = 0; i < N; ++i)
	{
		arr[i] = i;
	}

	arr[2] = 0;
	arr[5] = 0;
	arr[7] = 0;

	getSupportSet(sup, arr, N);

	for (int i = 0; i < N; ++i)
	{
		cout << i << " : " << sup[i] << endl;
	}

	getchar();
	delete[] arr;
	delete[] sup;
    return 0;
}