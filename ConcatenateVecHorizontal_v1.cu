#include "hip/hip_runtime.h"

#include "hip/hip_runtime_api.h"
#include "hipsolver.h"

#include <cstdio>
#include <cstdlib>
#include <iostream>

using namespace std;

int getMatrixRank(float *inmat,int M,int N,int *rank,float errthreshold)
{
	hipsolverHandle_t my_handle;
	hipsolverDnCreate(&my_handle);

	float *host_A = inmat;
	float *host_S = NULL;

	float *dev_A = NULL;
	float *dev_S = NULL;
	float *dev_U = NULL;
	float *dev_V = NULL;

	float *work = 0;
	int work_size = 0;
	int *dev_info = 0;

	host_S = new float[M*sizeof(float)];

	hipSetDevice(0);
	hipMalloc((void**)&dev_A, M * N * sizeof(float));
	hipMalloc((void**)&dev_S, M * sizeof(float));
	hipMalloc((void**)&dev_U, M * M * sizeof(float));
	hipMalloc((void**)&dev_V, N * N * sizeof(float));
	hipMemcpy(dev_A, host_A, M*N*sizeof(float), hipMemcpyHostToDevice);
	
	hipsolverDnSgesvd_bufferSize(my_handle, M, N, &work_size);
	hipMalloc((void**)&work, work_size * sizeof(float));
	hipMalloc((void**)&dev_info, sizeof(int));

	hipsolverDnSgesvd(my_handle, 'A', 'A', M, N, dev_A, M, dev_S, dev_U, M, dev_V, N, work, work_size, NULL, dev_info);
	hipDeviceSynchronize();

	hipMemcpy(host_S, dev_S, M*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dev_A);
	hipFree(dev_S);
	hipFree(dev_U);

	for (unsigned int i = 0; i < M; ++i)
	{
		if (*(host_S + i) > errthreshold) ++(*rank);
	}

	delete[] host_S;
	return 0;
}

int main()
{
	const int M = 3;
	const int N = 3;

	int rank = 0;

	cout << "Staring program...\n";

	float *host_A = NULL;
	host_A = new float[M*N*sizeof(float)];

	for (unsigned int i = 0; i < M; ++i)
	{
		for (unsigned int j = 0; j < N; ++j)
		{
			*(host_A + (i*M) + j) = i*j + 1;
		}
	}

	getMatrixRank(host_A, M, N, &rank, 0.001);

	cout << "Rank : " << rank << endl;

	getchar();
	hipDeviceReset();

    return 0;
}