#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hipsolver.h"
#include "hipblas.h"
#include "thrust\sort.h"
#include "thrust\device_new.h"
#include "thrust\device_vector.h"
#include "thrust\device_delete.h"
#include "thrust\host_vector.h"
#include "thrust\iterator\counting_iterator.h"
  
#include "AdditionalLibraries\readinput.h"
#include "AdditionalLibraries\CSVparser.hpp"
#include "AdditionalLibraries\CImg.h"

#include "SubFunctions\ConcatenateVecHorizontal.h"
#include "SubFunctions\getApproxBasis.h"
#include "SubFunctions\getMatrixRank.h"
#include "SubFunctions\getProjectionMatrix.h" // Change required
#include "SubFunctions\getPruneVector.h"
#include "SubFunctions\GetSupportSet.h"
#include "SubFunctions\getThresholdVector.h"
#include "SubFunctions\l1NormMinimization.h"
#include "SubFunctions\wl1NormMinimization.h"
#include "SubFunctions\LeastSquareEstimate.h"  //Change to new version
#include "SubFunctions\RecoverAndUpdate.h"

#include <iostream>
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <string>
//#include <Windows.h>

using namespace std;

// Actual Image Dimension 64x80 confirmed!!
// Total number of training images 1209
// Total number of measured images 1755

int showImage(thrust::device_vector<float> *imgdata, int M, int N, int rows, int cols)
{
	char c;
	cimg_library::CImg<unsigned char> image(cols,rows, 1, 1, 255);
	
	for (int i = 0; i < M; ++i)
	{
		image(i / rows, i % rows, 0) = (unsigned char)((*imgdata)[i]);
	}

	cimg_library::CImgDisplay main_disp(image, "MyImg", 0);

	//Sleep(2000);
	cin >> c;
	return 0;
}

int main()
{
	hipError_t err;                                                              //Error Status variable
	hipblasHandle_t bhandle;                                                       //CUBLAS Handle variable
	hipblasOperation_t opN = HIPBLAS_OP_N;                                          //CUBLAS Operation type
	hipblasOperation_t opT = HIPBLAS_OP_T;                                          //CUBLAS Operation type
	float alpha = 1, beta = 1, gamma = 1;// , one = 1, zero = 0;                  //CUBLAS Operation coefficient
//----------------------------------------------------------------------------------------------------------------------------------------------------------------	
	float *traindata = NULL;                                                      //Training Image Data
	float *imagedata = NULL;                                                      //Measured Image Data
	float *timagedata = NULL;                                                     //Transposed Image Data
	int Mtrain = 5120, Ntrain = 1755;                                             //Dimension of training image data
	int Mimg = 5120, Nimg = 1209;                                                 //Dimension of measured image data
	int M = 5120, N = 1209;                                                       //General dimensions -- live variables
	int rows = 64, cols = 80;                                                     //Image dimension
	int time = 0;                                                                 //Time count -- essentially number of frames left to process
	int d = 10;                                                                   //Frame interval
	int interval = 50;                                                            //Update interval
	char c;
//-----------------------------------------------------------------------------------------------------------------------------------------------------------------
	float b = 0.95;   //65                                                            //Fraction of total energy in approximate basis
	float threshold = 10;                                                      //Threshold for computing rank
	float omega = 0.1;                                                            //Initial omega value
	int rank = 0;                                                                 //Intial rank value
//-----------------------------------------------------------------------------------------------------------------------------------------------------------------
	float *dev_yt = NULL;
	float *dev_P0 = NULL;
	float *dev_Phit = NULL;
	float *dev_Mt = NULL;
	float *dev_St = NULL;
	float *dev_Stadd = NULL;
	float *dev_Stcap = NULL;
	float *dev_Lt1 = NULL;
	float *dev_Ltemp = NULL;
	float *dev_PLt = NULL;
	float epsilon = 0, lambda = 0;
//-----------------------------------------------------------------------------------------------------------------------------------------------------------------
	float *dev_traindata = NULL;                                                  //Memory pointer for training data
	float *dev_imgdata = NULL;                                                    //Memory pointer for image data  

	int *Told1 = new int[M];                                                      //Support Set for t-1 frame 
	int *Told2 = new int[M];                                                      //Suppost Set for t-2 frame
	unsigned int *Tadd = new unsigned int[M];                                     //Support Set for addtional frame
	unsigned int *T = new unsigned int[M];                                        //Support Set for running frame

	fill(Told1, Told1 + M, 0);
	fill(Told2, Told2 + M, 0);
//------------------------------------------------------------------------------------------------------------------------------------------------------------------	
	traindata = new float[Mtrain*Ntrain];                                         //Host memory for training data
	imagedata = new float[Mimg*Nimg];                                             //Host memory for image data
	timagedata = new float[Mimg*Nimg];                                            //Host memory for transposed image data, to access individual frames

	hipMalloc((void **)&dev_yt, M*sizeof(float));
	hipMalloc((void **)&dev_Phit, M*M*sizeof(float));
	hipMalloc((void **)&dev_Mt, M*sizeof(float));
	hipMalloc((void **)&dev_St, M*sizeof(float));
	hipMalloc((void **)&dev_Lt1, M*sizeof(float));
	hipMalloc((void **)&dev_Ltemp, M*sizeof(float));
	hipMalloc((void **)&dev_Stadd, M*sizeof(float));
	hipMalloc((void **)&dev_Stcap, M*sizeof(float));
	hipMalloc((void **)&dev_PLt, M*d*sizeof(float));
	
//-------------------------------------------------------------------------------------------------------------------------------------------------------------------	
	hipblasCreate(&bhandle);
	cout << "Reading input data...\n";
	ReadFile("curtaintraindata.csv", traindata, Mtrain, Ntrain); //5120x1755      Read training data
	cout << "Reading training data complete!\n";
	ReadFile("curtainimagedata.csv", imagedata, Mimg, Nimg); //5120x1209          Read image data
	cout << "Reading image data complete!\n";

//-------------------------------------------------------------------------------------------------------------------------------------------------------------------	
	cout << "Allocating device memory for data...\n";
	hipMalloc((void **)&dev_traindata, Mtrain*Ntrain*sizeof(float));                                //Allocate memory for training image data
	hipMalloc((void **)&dev_imgdata, Mimg*Nimg*sizeof(float));                                      //Allocate memory for actual image data
	cout << "Copying data to device memory...\n";
	hipMemcpy(dev_traindata, traindata, Mtrain*Ntrain*sizeof(float), hipMemcpyHostToDevice);       //Copy training data from host to device
	hipMemcpy(dev_imgdata, imagedata, Mimg*Nimg*sizeof(float), hipMemcpyHostToDevice);             //Copy image data from host to device
	
//-------------------------------------------------------------------------------------------------------------------------------------------------------------------
	time = 0;                                                                                        //Running time variable
	cout << "Computing initial approximate basis...\n";
	GetApproxBasis(dev_P0, &M, &N, dev_traindata, Mtrain, Ntrain, b);                                //Get the approximate basis
	
	cout << M << " " << N << endl;
	hipDeviceSynchronize();                                                                         //Synchronize
	cout << "Computing approximate basis rank...\n";
	
	getMatRank(&rank, dev_P0, threshold, M, N);                                                      //Get matrix rank of the basis
	thrust::device_vector<float> Mt(M);
	cout << rank << endl;
	hipDeviceSynchronize();                                                                         //Synchronize
	d = 10 * rank;                                                                                   //Set frame interval for subspace update
//-------------------------------------------------------------------------------------------------------------------------------------------------------------------
	cout << "Constructing approximate basis vector for runnig subspace...\n";
	thrust::device_vector<float> devPt(M*rank);                                                      //Construct running subspace basis
	thrust::device_vector<float> devPLt(M*d);                                                        //Construct running memory to store the previous dense vectors
	thrust::device_vector<float>::iterator it = devPt.begin();                                       //Get the iterator for the running subspace
	cout << "Initializing subspace vector array to zero values...\n";
	thrust::fill(devPLt.begin(), devPLt.end(), 0);                                                   //Initialize the running dense storage memory to zeros
	cout << "Initializing running approximate basis with computed approximate basis...\n";
	hipMemcpy(thrust::raw_pointer_cast(&devPt[0]), dev_P0, M*rank*sizeof(float), hipMemcpyDeviceToDevice); //Copy initial approximate basis data to running subspace basis
//-------------------------------------------------------------------------------------------------------------------------------------------------------------------
	cout << "Transposing measured image data for easy access...\n";
	hipblasSgeam(bhandle, opT, opN, Nimg, Mimg, &alpha, dev_imgdata, Mimg, &beta, NULL, Mimg, dev_imgdata, Nimg);     //Transpose the image data
	cout << "Constructing running measurement vector...\n";
	//thrust::device_vector<float> Mt(Mimg);                                                                     //Construct the running image vector
	cout << "measurement vector created!\n";
	thrust::device_vector<float>::iterator itmt = Mt.begin();                                               //Get the iterator for the running image vector
	thrust::device_vector<float> devimagedata(Mimg*Nimg);
	thrust::device_vector<float>::iterator itimgt = devimagedata.begin();
	devimagedata.insert(itimgt, dev_imgdata, dev_imgdata + Mimg*Nimg);
//-------------------------------------------------------------------------------------------------------------------------------------------------------------------
	cout << "Just for check, first frame of measured data...\n";
	
	for (int i = 0; i < Mimg; ++i)
	{
		Mt[i] = devimagedata[i*Nimg];
	}
	showImage(&Mt,Mimg,Nimg,rows,cols);
//-------------------------------------------------------------------------------------------------------------------------------------------------------------------	
//	!!!!!!!!!!!!!!!!!!!!!!!   Need to compute the support set first !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
	while (time < N)                                                                                                                       //Actual loop begins
	{
		itmt = Mt.begin();
		Mt.insert(itmt, itimgt + (time*M), itimgt + ((time + 1)*M) - 1);                                                                   //Get running frame from image data
		getProjectionMat(dev_Phit, thrust::raw_pointer_cast(&devPt[0]), M, N);                                                             //Get the projection matrix
		hipDeviceSynchronize();
        //Check dimension N changed to M
		hipblasSgemv(bhandle, opN, M, M, &alpha, dev_Phit, M, thrust::raw_pointer_cast(&Mt[0]), 1, &gamma, dev_yt, 1);                   //Get the projection
		hipDeviceSynchronize();
		memcpy((void*)Told2, (void*)Told1, M*sizeof(int));                                                                                 //Copy support set data
		//getSupportSet(Told1, dev_yt, M);                                                                                                 //Not this support set
		if (findIntersection(Told1, Told2, M) < 0.5)                                                                                       //If change is more 
		{                                                                                                                                  //Then normal l1 minmization
			hipblasSgemv(bhandle, opN, M, N, &alpha, dev_Phit, M, dev_Lt1, 1, &gamma, dev_Ltemp, 1);                                     //Vector product with projection matrix
			hipDeviceSynchronize();
			hipblasSnrm2(bhandle, M, dev_Ltemp, 1, &epsilon); //Check what it calculates                                                 //Calculate epsilon
			hipDeviceSynchronize();
			performBregmanIterRegularization(dev_St, dev_yt, dev_Phit, epsilon, M, N);                                                     //Perform l1 norm minimization recovery
			hipDeviceSynchronize();
			omega = computeOmega(dev_Mt, M);                                                                                               //Computer omega -- threshold
			Thresh(T, dev_St, M, omega);                                                                                                   //Perfrom thresholding and get the support set
			hipDeviceSynchronize();
		}
		else                                                                                                                               //Else weighted l1 minimization
		{
			lambda = getSetDifference(Told1, Told2, M);                                                                                    //Compute the set difference fraction
			hipblasSgemv(bhandle, opN, M, N, &alpha, dev_Phit, M, dev_Lt1, 1, &gamma, dev_Ltemp, 1);                                     //Vector product with projection matrix
			hipDeviceSynchronize();                                                             
			hipblasSnrm2(bhandle, M, dev_Ltemp, 1, &epsilon);                                                                            //Calculate epsilon
			hipDeviceSynchronize();
			performWtdBregmanIterRegularization(dev_St, dev_yt, dev_Phit, Told1, lambda, epsilon, M, N);                                   //Perform wighted l1 norm minimization recovery                     
			hipDeviceSynchronize();
			Prune(Tadd, dev_St, M, 1.4*getSupportCardinality(Told1, M));                                                                   //Perform prunning to get support set
			hipDeviceSynchronize();
			getLeastSquareEstimate(dev_Stadd, dev_yt, dev_Phit, Tadd, M, N);                                                               //Get least square estimate of St
			hipDeviceSynchronize();
			omega = computeOmega(dev_Mt, M);                                                                                               //Computer omega -- threshold
			Thresh(T, dev_Stadd, M, omega);                                                                                                //Perfrom thresholding and get the support set
			hipDeviceSynchronize();
		}
		
		getLeastSquareEstimate(dev_Stcap, dev_yt, dev_Phit, T, M, N);                                                                      //Get final least square estimate of St 
		hipDeviceSynchronize();
		computeLt(dev_Lt1, dev_Mt, dev_Stcap, M);                                                                                          //Compute Lt
		
		bool dosvd = (time%interval)?false:true;                                                                                           //Check whether subspace needs to be updated
		UpdateSubspace(&devPt, dev_Lt1, &devPLt, M, N, d, rank, dosvd);                                                                    //Update the subspace
		hipDeviceSynchronize();
		time++;                                                                                                                            //Increase the time count
	}

	cin >> c;

	hipblasDestroy(bhandle);
	//hipsolverDnDestroy(my_handle);

	hipFree(dev_yt);
	hipFree(dev_Phit);
	hipFree(dev_Mt);
	hipFree(dev_Lt1);
	hipFree(dev_Ltemp);
	hipFree(dev_St);
	hipFree(dev_Stadd);
	hipFree(dev_Stcap);
	hipFree(dev_PLt);
	hipFree(dev_traindata);
	hipFree(dev_imgdata);

	delete[] traindata;
	delete[] imagedata;
	delete[] timagedata;
	delete[] Told1;
	delete[] Told2;
	delete[] Tadd;
	delete[] T;

	err = hipDeviceReset();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}
